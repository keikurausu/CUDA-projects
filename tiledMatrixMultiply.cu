#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

// Compute C = A * B
template<int TILE_WIDTH>
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
                                     int numAColumns, int numBRows,
                                     int numBColumns, int numCRows,
                                     int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int Row = by*blockDim.y + ty;
	int Col = bx*blockDim.x + tx;
	float Cvalue = 0;
	
	for(int t = 0; t<(numBRows-1)/TILE_WIDTH+1; t++)
	{
		if(Row < numARows && (t*TILE_WIDTH + tx < numAColumns))
		{
			ds_A[ty][tx] = A[Row*numBRows + t*TILE_WIDTH + tx];
		}
		else
		{
			ds_A[ty][tx] = 0.0;
		}
		
		if(Col < numBColumns && (t*TILE_WIDTH + ty < numBRows))
		{
			ds_B[ty][tx] = B[(t*TILE_WIDTH + ty)*numBColumns + Col];
		}
		else
		{
			ds_B[ty][tx] = 0.0;
		}

		__syncthreads();
		
		for(int i = 0; i<TILE_WIDTH; i++)
		{
			Cvalue += ds_A[ty][i] * ds_B[i][tx];	
		}
		__syncthreads();
	}
	if(Row < numCRows && Col < numCColumns)
	{
		C[Row*numCColumns + Col] = Cvalue;
	}
	
	
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set this)

  args = wbArg_read(argc, argv);
	
	

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =
      ( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =
      ( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  wbCheck(hipHostMalloc(&hostC, (numCRows*numCColumns*sizeof(float))));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  wbCheck(hipMalloc(&deviceA, (numARows*numAColumns*sizeof(float))));
  wbCheck(hipMalloc(&deviceB, (numBRows*numBColumns*sizeof(float))));
  wbCheck(hipMalloc(&deviceC, (numCRows*numCColumns*sizeof(float))));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  wbCheck(hipMemcpy(deviceA, hostA, (numARows*numAColumns*sizeof(float)), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceB, hostB, (numBRows*numBColumns*sizeof(float)), hipMemcpyHostToDevice));

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 grid(((numCColumns - 1) / 32 + 1), ((numCRows - 1) / 32 + 1));
  dim3 threads(32, 32);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<32> <<<grid, threads >>>(deviceA, deviceB, deviceC,
	  numARows, numAColumns,
	  numBRows, numBColumns,
	  numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  wbCheck(hipMemcpy(hostC, deviceC, (numCRows*numCColumns*sizeof(float)), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  wbCheck(hipFree(deviceA));
  wbCheck(hipFree(deviceB));
  wbCheck(hipFree(deviceC));

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
	
  free(hostB);
	
  hipHostFree(hostC);
	printf("Hello");

  return 0;
}
